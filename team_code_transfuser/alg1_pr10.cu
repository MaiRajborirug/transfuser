
#include <hip/hip_runtime.h>

#define PI 3.141592654f
#define V_I_MIN 0.0f 
#define V_I_MAX 0.0f  // V 4.0

#define A_MIN 0.0f // A 1.0
#define A_MAX 0.0f

#define PSI_I_MIN 0.0f // 1.0
#define PSI_I_MAX 0.0f

// radius to keep obstacles out of, in meters
#define R 1.5f
#define ALPHA1 5.0f
#define ALPHA2 10.0f

// height threshold for obstacle avoidance
// this is down positive, origin at camera location
#define H_BAR 0.1f

#define IMG_W 960
#define IMG_H 480

#define Y_MIN 100.0f // Y > Y_MIN for target following 0.01
#define X_RANGE 50.0f // X < -X_RANGE or X > X_RANGE for target following 0.004
#define OMEGA_MIN 0.5f // stronger bound for omega (discrete doesn't suddenly change Omega)
#define MU_B 0.023f // 0.025
#define NU_B -0.00f //-0.03
#define MU_DOT_B 0.0f

#define H_MIN 2.1f
#define H_MAX 2.3f


__device__
float d_min(
    float f,
    float gamma_i,
    float Y_i
)
{
    float d_min = f * gamma_i / Y_i * H_MIN;
    return d_min;
}

__device__
float d_max(
    float f,
    float gamma_i,
    float Y_i
)
{
    float d_max = f * gamma_i / Y_i * H_MAX;
    return d_max;
}

// __device__
// float compute_h2(
//     float f,
//     float X_i,
//     float Y_i,
//     float gamma_i,
//     float d_i,
//     float v_e,
//     float a_e,
//     float omega_e,
//     float a_g,
//     float v_g,
//     float omega_g,
//     float is_animated,
//     float theta,
// )
// {
//     //eqn 28 get h2
//     float term2 = v_e * X_i / f / gamma_i * omega_e;
//     float term5 = -(a_e + ALPHA2 * v_e)/ gamma_i;
//     float term6 = ALPHA1 * ALPHA2 * (d_i - R);

//     if (is_animated){
//         float term1 = - V_I_MAX * PSI_I_MAX * sinf(theta);
//         float term3 = 1/d_i * (v)
//     }
//     else{ 
//         float h2 = term2 + term5 + term6;
//         return float(h2 >= 0);
//     }
// }

__device__
bool condition_i(
    float f, 
    float X_i, 
    float Y_i, 
    float v_e, 
    float a_e,
    float omega_e,   // Assuming psi_e should be omega_e
    int8_t is_animated,
    int8_t is_obstacle,
    int8_t is_wp,
    float d_upper_bound,
    float d_lower_bound) {  // Fixed extra comma
    // compute gamma
    float gamma_i = sqrtf(((X_i * X_i) / (f * f)) + 1.0f);
    
    // not region of interest
    if (Y_i < Y_MIN) {
        return true;
    }

    // target following
    else if (is_wp) {
        // animated object has more terms
        float term_vg = 0;
        if (is_animated) {
            term_vg = V_I_MAX / d_lower_bound;
        }

        // based on location
        if (X_i < X_RANGE && X_i > -X_RANGE) {
            return true;
        } else {
            float term_ve = v_e * X_i / f / gamma_i / d_lower_bound;
            
            // term: omega_e > OMEGA_MIN is the extra term to make sure that 
            // object changes angular velocity
            if (X_i > 0.0 && omega_e > term_vg && omega_e > OMEGA_MIN)
            {
                return true;
            }
            else if (X_i < 0.0 && omega_e < term_vg + term_ve && omega_e < -OMEGA_MIN)  // eqn 20
            {
                return true;
            }
            return false;
        }
    }

    // object avoidance
    else if (is_obstacle) {
        float term1 = 0.0f;
        float term2 = v_e * X_i / f / gamma_i * omega_e;
        float term3 = 1.0f / d_upper_bound * (v_e * X_i / f / gamma_i) * (v_e * X_i / f / gamma_i);
        float term4 = 0.0f;
        float term5 = -(a_e + ALPHA2 * v_e) / gamma_i;
        float term6 = ALPHA1 * ALPHA2 * (d_lower_bound - R);

        if (is_animated) {
            term1 = -V_I_MAX * PSI_I_MAX;
            term3 = 0.0f;
            term4 = (V_I_MAX + ALPHA2 * A_MAX);
        }

        float h2 = term1 + term2 + term3 + term4 + term5 + term6;
        return true;  // h2 is not used, so the logic is simplified
    }

    // not object of interest
    else {
        return true;
    }
}


__global__
void certify_u(
    float f, 
    // float* mu_is,
    // float* nu_is, 
    float* X_i_ts, 
    float* Y_i_ts, 
    // float* offsets,
    float v_e, 
    float psi_e, 
    float a_e, 
    // float phi_e, 
    // unsigned int N_points,
    // unsigned int N_pixels,
    bool* u_certifieds,
    // float* nu_b_out,
    // float* nu_i_out,
    // float* nu_dot_out,
    int8_t* animateds,
    int8_t* wps,
    int8_t* obstacles,
    float* d_upper_bounds,
    float* d_lower_bounds
)
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int tid = blockId * (blockDim.x * blockDim.y)
        + (threadIdx.y * blockDim.x) + threadIdx.x;

    
    if (tid >= IMG_H * IMG_W){
        return;
    } 
    float X_i_t = X_i_ts[tid];
    float Y_i_t = Y_i_ts[tid];
    int8_t is_animated = animateds[tid];
    int8_t is_wp = wps[tid];
    int8_t is_obstacle = obstacles[tid];
    float gamma_i =  sqrtf(((X_i_t * X_i_t) / (f * f)) + 1.0f);
    
    float d_upper_bound;
    float d_lower_bound;
    
    // d_upper_bound = d_upper_bounds[tid];
    // d_lower_bound = d_lower_bounds[tid];

    d_upper_bound = d_max(f, gamma_i, Y_i_t);
    d_lower_bound = d_min(f, gamma_i, Y_i_t);

    bool cert_tid = condition_i(
        f, 
        X_i_t, 
        Y_i_t, 
        v_e, 
        a_e,
        psi_e, // If psi_e is actually omega_e
        is_animated,
        is_obstacle,
        is_wp,
        d_upper_bound,
        d_lower_bound);
    u_certifieds[tid] = cert_tid;
    // int a = 1;
}




// //-----
// __device__ 
// float D_mu(
//     float f, 
//     float mu_i, 
//     float X_i_t,
//     float v_i,
//     float v_e,
//     float theta, 
//     float psi_e, 
//     float CHI)
// {
//     float nom = 
//     f * (X_i_t * v_e - X_i_t * v_i * cosf(theta) + f * v_i * sinf(theta));
//     float denom = 
//     CHI * (psi_e * X_i_t * X_i_t + psi_e * f * f - mu_i * f);
//     return nom / denom;
// }

// __device__         
// void solve_eqn_43_for_theta(
//     float f, 
//     float X_i_t, 
//     float* D_theta_candidates
// ){
//     float theta = atan2f(-f, X_i_t);
//     D_theta_candidates[0] = theta;
//     D_theta_candidates[1] = fmodf(theta + PI, 2 * PI);
// }

// __device__
// void optimize_D_mu(
//     float f, 
//     float mu_i, 
//     float X_i_t, 
//     float v_e, 
//     float psi_e, 
//     float CHI,
//     float* D_max,
//     float* D_min,
//     int8_t animated
// ){
//     float D_theta_candidates[2];
//     solve_eqn_43_for_theta(f, X_i_t, D_theta_candidates);
//     // V_I_MIN must be zero
//     float candidates[3];

//     // D_mu is constant w.r.t theta when v = 0

//     candidates[0] = D_mu(
//         f, 
//         mu_i, 
//         X_i_t, 
//         V_I_MIN * animated,
//         v_e,
//         D_theta_candidates[0],
//         psi_e, 
//         CHI);

//     candidates[1] = D_mu(
//         f, 
//         mu_i, 
//         X_i_t, 
//         V_I_MAX * animated,
//         v_e,
//         D_theta_candidates[0],
//         psi_e, 
//         CHI);

//     candidates[2] = D_mu(
//         f, 
//         mu_i, 
//         X_i_t, 
//         V_I_MAX * animated,
//         v_e,
//         D_theta_candidates[1],
//         psi_e, 
//         CHI);

    
//     bool first_iter = true;
//     float max;
//     float min;
//     for (int i = 0; i < 3; i++){
//         if (first_iter){
//             max = candidates[i];
//             min = candidates[i];
//             first_iter = false;
//         } else {
//             if (candidates[i] > max){
//                 max = candidates[i];
//             } else if (candidates[i] < min) {
//                 min = candidates[i];
//             }
//         }
//     }

//     //*D_max = max;
//     if (max > R){
//         *D_max = max;
//     } else {
//         *D_max = R;
//     }
//     if (min > R){
//         *D_min = min;
//     } else {
//         *D_min = R;
//     }
// }

// __device__
// float D_nu(
//     float f, 
//     float nu_i, 
//     float X_i_t,
//     float Y_i_t, 
//     float v_i,
//     float v_e,
//     float theta, 
//     float psi_e, 
//     float CHI)
// {   
//     float nom =
//     Y_i_t * f * (v_e - v_i * cosf(theta));
//     float denom = 
//     CHI * (f * nu_i - X_i_t * Y_i_t * psi_e);
//     return nom / denom;
// }

// __device__ 
// void optimize_D_nu(
//     float f, 
//     float nu_i, 
//     float X_i_t, 
//     float Y_i_t,
//     float v_e, 
//     float psi_e, 
//     float CHI,
//     float* D_max,
//     float* D_min,
//     int8_t animated
// ){
//     float theta_candidate_1 = 0.0f;
//     float theta_candidate_2 = PI;
//     float candidates[3];

//     candidates[0] = D_nu(
//         f, 
//         nu_i, 
//         X_i_t,
//         Y_i_t, 
//         0.0f,
//         v_e,
//         theta_candidate_1, 
//         psi_e, 
//         CHI);

//     candidates[1] = D_nu(
//         f, 
//         nu_i, 
//         X_i_t,
//         Y_i_t, 
//         V_I_MAX * animated,
//         v_e,
//         theta_candidate_1, 
//         psi_e, 
//         CHI);

//     candidates[2] = D_nu(
//         f, 
//         nu_i, 
//         X_i_t,
//         Y_i_t, 
//         V_I_MAX * animated,
//         v_e,
//         theta_candidate_2, 
//         psi_e, 
//         CHI);

//     bool first_iter = true;
//     float max;
//     float min;
//     for (int i = 0; i < 3; i++){
//         if (first_iter){
//             max = candidates[i];
//             min = candidates[i];
//             first_iter = false;
//         } else {
//             if (candidates[i] > max){
//                 max = candidates[i];
//             } else if (candidates[i] < min) {
//                 min = candidates[i];
//             }
//         }
//     }

//     //*D_max = max;
//     if (max > R){
//         *D_max = max;
//     } else {
//         *D_max = R;
//     }
//     if (min > R){
//         *D_min = min;
//     } else {
//         *D_min = R;
//     }
// }

// __device__
// void calculate_D_bounds(
//     float f, 
//     float mu_i,        
//     float nu_i, 
//     float X_i_t,
//     float Y_i_t, 
//     float v_e, 
//     float psi_e, 
//     float CHI,
//     float* D_min,
//     float* D_max,
//     int8_t animated
//     )
// {
//     float D_min_candidate_mu = 0.0f;
//     float D_max_candidate_mu = 0.0f;
//     float D_min_candidate_nu = 0.0f;
//     float D_max_candidate_nu = 0.0f;

//     optimize_D_mu(
//         f, 
//         mu_i, 
//         X_i_t, 
//         v_e, 
//         psi_e, 
//         CHI,
//         &D_max_candidate_mu,
//         &D_min_candidate_mu,
//         animated
//     );

//     optimize_D_nu(
//         f, 
//         nu_i, 
//         X_i_t, 
//         Y_i_t,
//         v_e, 
//         psi_e, 
//         CHI,
//         &D_max_candidate_nu,
//         &D_min_candidate_nu,
//         animated
//     );

//     *D_min = fmaxf(D_min_candidate_mu, D_min_candidate_nu);
//     *D_max = fminf(D_max_candidate_mu, D_max_candidate_nu);
// }


// __device__
// float eqn_54(float a_i,
//             float f, 
//             float mu_i, 
//             float nu_i, 
//             float X_i_t, 
//             float Y_i_t, 
//             float v_i, 
//             float v_e, 
//             float psi_i,
//             float psi_e, 
//             float a_e,
//             float theta, 
//             float phi_e, 
//             float CHI, 
//             float D)
// {
//     float term1 = phi_e * f;
//     float term2 = 2 * mu_i * nu_i / Y_i_t;
//     float term3 = (X_i_t * X_i_t) * phi_e / f;
//     float D_CHI = D * CHI;
//     float term4 = X_i_t * a_e;
//     float term5 = a_i * f * sinf(theta); 
//     float term6 = -f * psi_e * v_e;
//     float COS_THETA = cosf(theta);
//     float term7 = -X_i_t * a_i * COS_THETA;
//     float term8 = 2 * f * psi_e * v_i * COS_THETA;
//     float term9 = -f * psi_i * v_i * COS_THETA;
//     float term10 = sinf(theta) * (2 * X_i_t * psi_e * v_i - X_i_t * psi_i * v_i);

//     return term1 + term2 + term3 + 
//         (term4 + term5 + term6 + term7 + term8 + term9 + term10) / D_CHI;
//     //return f;
// }









// // __device__
// // void solve_eqn_60_for_0(float f, float X_i_t, float* theta_candidates){
// //     float theta = atanf(X_i_t / f);
// //     theta_candidates[0] = theta;
// //     theta_candidates[1] = fmodf(theta + PI, 2 * PI);
// // }

// // __device__
// // void solve_theta_guess_from_59(float f, float X_i_t, float* theta_candidates){
// //     float theta = atanf(-f / X_i_t);
// //     theta_candidates[2] = theta;
// //     theta_candidates[3] = fmodf(theta + PI, 2 * PI);
// // }

// // __device__
// // void solve_eqn_59_for_0(float f, float X_i_t, float* theta_candidates){
// //     solve_theta_guess_from_59(f, X_i_t, theta_candidates);
// // }

// __device__
// float get_mu_dot(
//             float f, 
//             float x_v_i,
//             float y_v_i,
//             float x_a_i,
//             float y_a_i,
//             float X_i, 
//             float Y_i,
//             float mu_i, 
//             float nu_i,  
//             float v_e,
//             float a_e,
//             float omega_e,
//             float alpha_e,
//             float D_i,
//             float gamma_i)
// {
//     // float gamma_i =  sqrtf(((X_i * X_i) / (f * f)) + 1.0f);
//     float term1 = gamma_i * X_i / D_i * (-x_a_i - 2 * omega_e * y_v_i + a_e);
//     float term2 = gamma_i * f / D_i * (y_a_i + 2 * omega_e * x_v_i + omega_e * v_e);
//     float term3 = - gamma_i * gamma_i * alpha_e;
//     float term4 = 2*mu_i*nu_i/Y_i;
//     return term1 + term2 + term3 + term4;
// }

// __device__
// float optimize_mu_dot_i2(
//             float f, 
//             float X_i, 
//             float Y_i, 
//             float mu_i,
//             float nu_i, 
//             float v_e, 
//             float a_e,
//             float omega_e,
//             float alpha_e,
//             // unsigned int N,
//             bool findmax,
//             int8_t animated,
//             float d_upper_bound,
//             float d_lower_bound)
// {
//     // iterate over boundary values of x_v_i, y_v_i, x_a_i, y_a_i, D_i

//     // compute gamma
//     float gamma_i =  sqrtf(((X_i * X_i) / (f * f)) + 1.0f);

//     // float D_MAX = 0.0f;
//     // float D_MIN = 0.0f;
//     // calculate_D_bounds(
//     //     f, 
//     //     mu_i,        
//     //     nu_i, 
//     //     X_i_t,
//     //     Y_i_t, 
//     //     v_e, 
//     //     psi_e, 
//     //     CHI,
//     //     &D_MIN,
//     //     &D_MAX,
//     //     animated); // assign D_MIN, D_MAX new values
//     float x_v_i_candidates[2] = {-V_I_MAX, V_I_MAX};
//     float y_v_i_candidates[2] = {-V_I_MAX, V_I_MAX};
//     float x_a_i_candidates[2] = {-A_MAX, A_MAX};
//     float y_a_i_candidates[2] = {-A_MAX, A_MAX};
//     float D_candidates[2] ={d_upper_bound, d_lower_bound};

//     // best_y = mu_dot or nu_dot output
//     float best_y = 0.0f;
//     bool first_iter = true; 
//     float candidate_y;

//     // non animate and animate objects
//     if (animated){
//         for (int x_v_i_idx = 0; x_v_i_idx < 2; x_v_i_idx ++){
//             for (int y_v_i_idx = 0; y_v_i_idx < 2; y_v_i_idx ++){
//                 for (int x_a_i_idx = 0; x_a_i_idx < 2; x_a_i_idx ++){
//                     for (int y_a_i_idx = 0; y_a_i_idx < 2; y_a_i_idx ++){
//                         for (int D_idx = 0; D_idx < 2; D_idx ++){
//                             float x_v_i_candidate = x_v_i_candidates[x_v_i_idx];
//                             float y_v_i_candidate = y_v_i_candidates[y_v_i_idx];
//                             float x_a_i_candidate = x_a_i_candidates[x_a_i_idx];
//                             float y_a_i_candidate = y_a_i_candidates[y_a_i_idx];
//                             float D_candidate = D_candidates[D_idx];

//                             candidate_y = get_mu_dot(
//                                 f, 
//                                 x_v_i_candidate,
//                                 y_v_i_candidate,
//                                 x_a_i_candidate,
//                                 y_a_i_candidate,
//                                 X_i, 
//                                 Y_i,
//                                 mu_i, 
//                                 nu_i,  
//                                 v_e,
//                                 a_e,
//                                 omega_e,
//                                 alpha_e,
//                                 D_candidate,
//                                 gamma_i);

//                             if (first_iter){
//                                 best_y = candidate_y;
//                                 first_iter = false;
//                             }
//                             else if (findmax && best_y < candidate_y){
//                                 best_y = candidate_y;
//                             }else if (!findmax && best_y > candidate_y){
//                                 best_y = candidate_y;
//                             }
//                         }
//                     }
//                 }
//             }
//         }
//     }
//     else {
//         for (int D_idx = 0; D_idx < 2; D_idx ++){
//             float x_v_i_candidate = 0.0;
//             float y_v_i_candidate = 0.0;
//             float x_a_i_candidate = 0.0;
//             float y_a_i_candidate = 0.0;
//             float D_candidate = D_candidates[D_idx];
            
//             candidate_y = get_mu_dot(
//                 f, 
//                 x_v_i_candidate,
//                 y_v_i_candidate,
//                 x_a_i_candidate,
//                 y_a_i_candidate,
//                 X_i, 
//                 Y_i,
//                 mu_i, 
//                 nu_i,  
//                 v_e,
//                 a_e,
//                 omega_e,
//                 alpha_e,
//                 D_candidate,
//                 gamma_i);

//             if (first_iter){
//                 best_y = candidate_y;
//                 first_iter = false;
//             }
//             else if (findmax && best_y < candidate_y){
//                 best_y = candidate_y;
//             }else if (!findmax && best_y > candidate_y){
//                 best_y = candidate_y;
//             }
//         }
//     }
//     return best_y;
//     //return D_MAX;
// }

// // __device__
// // float optimize_mu_dot_i(
// //             float f, 
// //             float mu_i,
// //             float nu_i, 
// //             float X_i_t, 
// //             float Y_i_t, 
// //             float v_e, 
// //             float psi_e, 
// //             float a_e, 
// //             float phi_e,
// //             unsigned int N,
// //             bool findmax,
// //             int8_t animated,
// //             float d_upper_bound,
// //             float d_lower_bound)
// // {
// //     // iterate over boundary values of a_i, v_i, psi_i and D,
// //     // brute force over theta

// //     // compute CHI
// //     // inverse squre root
// //     float CHI = rsqrtf(((X_i_t * X_i_t) / (f * f)) + 1.0f);

// //     // float D_MAX = 0.0f;
// //     // float D_MIN = 0.0f;
// //     // calculate_D_bounds(
// //     //     f, 
// //     //     mu_i,        
// //     //     nu_i, 
// //     //     X_i_t,
// //     //     Y_i_t, 
// //     //     v_e, 
// //     //     psi_e, 
// //     //     CHI,
// //     //     &D_MIN,
// //     //     &D_MAX,
// //     //     animated); // assign D_MIN, D_MAX new values
// //     float a_i_candidates[2] = {A_MIN, A_MAX};
// //     float v_i_candidates[2] = {V_I_MIN, V_I_MAX};
// //     float psi_i_candidates[2] = {PSI_I_MIN, PSI_I_MAX};
// //     float D_candidates[2] ={d_upper_bound, d_lower_bound};
// //     float theta_candidate = 0.0f;
// //     float theta_stepsize = 2.0f * PI / (float)N;

// //     float best_y = 0.0f;
// //     bool first_iter = true; 
// //     float candidate_y;

// //     // non animate and animate objects
// //     if (animated){
// //         for (int a_i_idx = 0; a_i_idx < 2; a_i_idx ++){
// //             for (int v_i_idx = 0; v_i_idx < 2; v_i_idx ++){
// //                 for (int psi_i_idx = 0; psi_i_idx < 2; psi_i_idx ++){
// //                     for (int D_idx = 0; D_idx < 2; D_idx ++){
// //                         float a_i_candidate = a_i_candidates[a_i_idx];
// //                         float v_i_candidate = v_i_candidates[v_i_idx];
// //                         float psi_i_candidate = psi_i_candidates[psi_i_idx];
// //                         float D_candidate = D_candidates[D_idx];

// //                         for (int i = 0; i < N; i++){
// //                             candidate_y = eqn_54(a_i_candidate,
// //                                             f, 
// //                                             mu_i, 
// //                                             nu_i, 
// //                                             X_i_t,
// //                                             Y_i_t,
// //                                             v_i_candidate, 
// //                                             v_e, 
// //                                             psi_i_candidate,
// //                                             psi_e,
// //                                             a_e,
// //                                             theta_candidate,
// //                                             phi_e,
// //                                             CHI, 
// //                                             D_candidate);

// //                             if (first_iter){
// //                                 best_y = candidate_y;
// //                                 first_iter = false;
// //                             }
// //                             else if (findmax && best_y < candidate_y){
// //                                 best_y = candidate_y;
// //                             }else if (!findmax && best_y > candidate_y){
// //                                 best_y = candidate_y;
// //                             }
// //                             theta_candidate += theta_stepsize;
// //                         }
// //                     }
// //                 }
// //             }
// //         }
// //     }
// //     else {
// //         // for (int D_idx = 0; D_idx < 2; D_idx ++){
// //             // float D_candidate = D_candidates[D_idx];
// //         float D_candidate = d_upper_bound;

// //         for (int i = 0; i < N; i++){
// //             candidate_y = eqn_54(0.0f, //a_i_candidate,
// //                             f, 
// //                             mu_i, 
// //                             nu_i, 
// //                             X_i_t,
// //                             Y_i_t,
// //                             0.0f, //v_i_candidate, 
// //                             v_e, 
// //                             0.0f, //psi_i_candidate,
// //                             psi_e,
// //                             a_e,
// //                             theta_candidate,
// //                             phi_e,
// //                             CHI, 
// //                             D_candidate);

// //             if (first_iter){
// //                 best_y = candidate_y;
// //                 first_iter = false;
// //             }
// //             else if (findmax && best_y < candidate_y){
// //                 best_y = candidate_y;
// //             }else if (!findmax && best_y > candidate_y){
// //                 best_y = candidate_y;
// //             }
// //             theta_candidate += theta_stepsize;
// //         }
// //         // }
// //     }
// //     return best_y;
// //     //return D_MAX;
// // }

// __global__
// void certify_u_for_mu(
//             float f, 
//             float* X_i_ts, 
//             float* Y_i_ts,
//             float* mu_is,
//             float* nu_is, 
//             float* offsets,
//             float v_e, 
//             float a_e,
//             float omega_e, 
//             float alpha_e, 
//             unsigned int N_points,
//             unsigned int N_pixels,
//             bool* u_certified_for_mu,
//             float* mu_b_out,
//             float* mu_i_out,
//             float* mu_dot_out,
//             int8_t* animateds,
//             int8_t* rois, // area of interest?
//             float* d_upper_bounds,
//             float* d_lower_bounds){

//     int blockId = blockIdx.x + blockIdx.y * gridDim.x;
//     int tid = blockId * (blockDim.x * blockDim.y)
//         + (threadIdx.y * blockDim.x) + threadIdx.x;
    
//     if (tid >= IMG_H * IMG_W){
//         return;
//     }

//     float X_i_t = X_i_ts[tid];
//     float Y_i_t = Y_i_ts[tid];
//     float mu_i = mu_is[tid];
//     float nu_i = nu_is[tid]; 
//     float d_upper_bound = d_upper_bounds[tid];
//     float d_lower_bound = d_lower_bounds[tid];
//     float offset = offsets[tid];
//     int8_t animated = animateds[tid];
//     int8_t roi = rois[tid];


//     mu_i_out[tid] = mu_i;
//     bool nu_find_upperbound = Y_i_t > Y_MIN;
//     bool mu_find_left = X_i_t <= -X_RANGE; // check whimsicle
//     bool mu_find_right = X_i_t >= X_RANGE;
//     bool mu_find_mid = (X_i_t <= X_RANGE) && (X_i_t >= -X_RANGE);

//     if (roi == 0){ // not a target -> skip computation
//         u_certified_for_mu[tid] = true;
//         mu_dot_out[tid] = 0.0f;
//         mu_b_out[tid] = 0.0f;
//         return;
//     }
//     else if (nu_find_upperbound && mu_find_right && mu_i >= -MU_B){ // right side event
//         mu_b_out[tid] = -MU_B;
//         float mu_upper = optimize_mu_dot_i2(
//             f, 
//             X_i_t, 
//             Y_i_t, 
//             mu_i,
//             nu_i, 
//             v_e, 
//             a_e,
//             omega_e,
//             alpha_e,
//             true,
//             animated,
//             d_upper_bound,
//             d_lower_bound);
//         u_certified_for_mu[tid] = mu_upper < MU_DOT_B;
//         mu_dot_out[tid] = mu_upper;
//     } else if (nu_find_upperbound && mu_find_left && mu_i <= MU_B){ // left side event
//         mu_b_out[tid] = MU_B;
//         float mu_lower = optimize_mu_dot_i2(
//             f, 
//             X_i_t, 
//             Y_i_t, 
//             mu_i,
//             nu_i, 
//             v_e, 
//             a_e,
//             omega_e,
//             alpha_e,
//             false,
//             animated,
//             d_upper_bound,
//             d_lower_bound);
//         u_certified_for_mu[tid] = mu_lower > MU_DOT_B; // 0.0f; =
//         mu_dot_out[tid] = mu_lower;
//     } else { // currently follow path
//         mu_b_out[tid] = 0.0f;
//         u_certified_for_mu[tid] = true;
//         mu_dot_out[tid] = 0.0f; 
//     }
// }














// __device__
// float eqn_55(float a_i,
//             float f, 
//             float mu_i, 
//             float nu_i, 
//             float X_i_t, 
//             float Y_i_t, 
//             float v_i, 
//             float v_e, 
//             float psi_i,
//             float psi_e, 
//             float a_e,
//             float theta, 
//             float phi_e, 
//             float CHI, 
//             float D)
// {
//     float term1 = Y_i_t * psi_e * psi_e;
//     float D_CHI = D * CHI;
//     float term2 = 2.0f * Y_i_t * v_i * sinf(theta) * psi_e / D_CHI;
//     float term3 = 2.0f * nu_i * nu_i / Y_i_t;
//     float term4 = Y_i_t * a_e / D_CHI;
//     float term5 = X_i_t * Y_i_t * phi_e / f;
//     float term6 = -Y_i_t * a_i * cosf(theta) / D_CHI;
//     float term7 = -Y_i_t * psi_i * v_i * sinf(theta) / D_CHI;
//     return term1 + term2 + term3 + term4 + term5 + term6 + term7;
//     //return term1;
// }

// __device__
// float get_nu_dot(
//             float f, 
//             float x_v_i,
//             float y_v_i,
//             float x_a_i,
//             float y_a_i,
//             float X_i, 
//             float Y_i,
//             float mu_i, 
//             float nu_i,  
//             float v_e,
//             float a_e,
//             float omega_e,
//             float alpha_e,
//             float D_i,
//             float gamma_i)
// {
//     // float gamma_i =  sqrtf(((X_i * X_i) / (f * f)) + 1.0f);
//     float term1 = gamma_i * Y_i / D_i * (-x_a_i - 2 * omega_e * y_v_i + a_e);
//     float term2 = omega_e * omega_e * Y_i;
//     float term3 = - alpha_e * X_i * Y_i / f;
//     float term4 = nu_i * nu_i / Y_i;
//     return term1 + term2 + term3 + term4;
// }

// __device__
// float optimize_nu_dot_i2(
//             float f, 
//             float X_i, 
//             float Y_i, 
//             float mu_i,
//             float nu_i, 
//             float v_e, 
//             float a_e,
//             float omega_e,
//             float alpha_e,
//             // unsigned int N,
//             bool findmax,
//             int8_t animated,
//             float d_upper_bound,
//             float d_lower_bound
// )
// {
//     // iterate over boundary values of x_v_i, y_v_i, x_a_i, y_a_i, D_i

//     // compute gamma
//     float gamma_i =  sqrtf(((X_i * X_i) / (f * f)) + 1.0f);

//     // float D_MAX = 0.0f;
//     // float D_MIN = 0.0f;
//     // calculate_D_bounds(
//     //     f, 
//     //     mu_i,        
//     //     nu_i, 
//     //     X_i_t,
//     //     Y_i_t, 
//     //     v_e, 
//     //     psi_e, 
//     //     CHI,
//     //     &D_MIN,
//     //     &D_MAX,
//     //     animated); // assign D_MIN, D_MAX new values
//     float x_v_i_candidates[2] = {-V_I_MAX, V_I_MAX};
//     float y_v_i_candidates[2] = {-V_I_MAX, V_I_MAX};
//     float x_a_i_candidates[2] = {-A_MAX, A_MAX};
//     float y_a_i_candidates[2] = {-A_MAX, A_MAX};
//     float D_candidates[2] ={d_upper_bound, d_lower_bound};

//     // best_y = mu_dot or nu_dot output
//     float best_y = 0.0f;
//     bool first_iter = true; 
//     float candidate_y;

//     // non animate and animate objects
//     if (animated){
//         for (int x_v_i_idx = 0; x_v_i_idx < 2; x_v_i_idx ++){
//             for (int y_v_i_idx = 0; y_v_i_idx < 2; y_v_i_idx ++){
//                 for (int x_a_i_idx = 0; x_a_i_idx < 2; x_a_i_idx ++){
//                     for (int y_a_i_idx = 0; y_a_i_idx < 2; y_a_i_idx ++){
//                         for (int D_idx = 0; D_idx < 2; D_idx ++){
//                             float x_v_i_candidate = x_v_i_candidates[x_v_i_idx];
//                             float y_v_i_candidate = y_v_i_candidates[y_v_i_idx];
//                             float x_a_i_candidate = x_a_i_candidates[x_a_i_idx];
//                             float y_a_i_candidate = y_a_i_candidates[y_a_i_idx];
//                             float D_candidate = D_candidates[D_idx];

//                             candidate_y = get_nu_dot(
//                                 f, 
//                                 x_v_i_candidate,
//                                 y_v_i_candidate,
//                                 x_a_i_candidate,
//                                 y_a_i_candidate,
//                                 X_i, 
//                                 Y_i,
//                                 mu_i, 
//                                 nu_i,  
//                                 v_e,
//                                 a_e,
//                                 omega_e,
//                                 alpha_e,
//                                 D_candidate,
//                                 gamma_i);

//                             if (first_iter){
//                                 best_y = candidate_y;
//                                 first_iter = false;
//                             }
//                             else if (findmax && best_y < candidate_y){
//                                 best_y = candidate_y;
//                             }else if (!findmax && best_y > candidate_y){
//                                 best_y = candidate_y;
//                             }
//                         }
//                     }
//                 }
//             }
//         }
//     }
//     else {
//         for (int D_idx = 0; D_idx < 2; D_idx ++){
//             float x_v_i_candidate = 0.0;
//             float y_v_i_candidate = 0.0;
//             float x_a_i_candidate = 0.0;
//             float y_a_i_candidate = 0.0;
//             float D_candidate = D_candidates[D_idx];
            
//             candidate_y = get_nu_dot(
//                 f, 
//                 x_v_i_candidate,
//                 y_v_i_candidate,
//                 x_a_i_candidate,
//                 y_a_i_candidate,
//                 X_i, 
//                 Y_i,
//                 mu_i, 
//                 nu_i,  
//                 v_e,
//                 a_e,
//                 omega_e,
//                 alpha_e,
//                 D_candidate,
//                 gamma_i);

//             if (first_iter){
//                 best_y = candidate_y;
//                 first_iter = false;
//             }
//             else if (findmax && best_y < candidate_y){
//                 best_y = candidate_y;
//             }else if (!findmax && best_y > candidate_y){
//                 best_y = candidate_y;
//             }
//         }
//     }
//     return best_y;
//     //return D_MAX;
// }


// // __device__ 
// // float optimize_nu_dot_i(
// //             float f, 
// //             float mu_i,
// //             float nu_i, 
// //             float X_i_t, 
// //             float Y_i_t, 
// //             float v_e, 
// //             float psi_e, 
// //             float a_e, 
// //             float phi_e, 
// //             unsigned int N, 
// //             bool findmax,
// //             int8_t animated,
// //             float d_upper_bound,
// //             float d_lower_bound){

// //     // brute force over theta,
// //     // use boundary values for a_i, v_i, psi_e and D
// //     // since mu_dot is monotonomus w.r.t these 4 variables

    
// //      // iterate over boundary values of a_i, v_i, psi_i and D,
// //     // brute force over theta

// //     // compute CHI
// //     // inverse squre root

// //     float CHI = rsqrtf(((X_i_t * X_i_t) / (f * f)) + 1.0f);

// //     // float D_MIN = 0.0f;
// //     // float D_MAX = d_depth_array;
// //     // calculate_D_bounds(
// //     //     f, 
// //     //     mu_i,        
// //     //     nu_i, 
// //     //     X_i_t,
// //     //     Y_i_t, 
// //     //     v_e, 
// //     //     psi_e, 
// //     //     CHI,
// //     //     &D_MIN,
// //     //     &D_MAX,
// //     //     animated);

// //     float a_i_candidates[2] = {A_MIN, A_MAX};
// //     float v_i_candidates[2] = {V_I_MIN, V_I_MAX};
// //     float psi_i_candidates[2] = {PSI_I_MIN, PSI_I_MAX};
// //     float D_candidates[2] ={d_upper_bound, d_lower_bound};
// //     float theta_candidate = 0.0f;
// //     float theta_stepsize = 2.0f * PI / (float)N;

// //     float best_y = 0.0f;
// //     bool first_iter = true; 
// //     float candidate_y;

// //     // non animate and animate objects
// //     if (animated){
// //         for (int a_i_idx = 0; a_i_idx < 2; a_i_idx ++){
// //             for (int v_i_idx = 0; v_i_idx < 2; v_i_idx ++){
// //                 for (int psi_i_idx = 0; psi_i_idx < 2; psi_i_idx ++){
// //                     for (int D_idx = 0; D_idx < 2; D_idx ++){
// //                         float a_i_candidate = a_i_candidates[a_i_idx];
// //                         float v_i_candidate = v_i_candidates[v_i_idx];
// //                         float psi_i_candidate = psi_i_candidates[psi_i_idx];
// //                         float D_candidate = D_candidates[D_idx];

// //                         for (int i = 0; i < N; i++){
// //                             candidate_y = eqn_54(a_i_candidate,
// //                                             f, 
// //                                             mu_i, 
// //                                             nu_i, 
// //                                             X_i_t,
// //                                             Y_i_t,
// //                                             v_i_candidate, 
// //                                             v_e, 
// //                                             psi_i_candidate,
// //                                             psi_e,
// //                                             a_e,
// //                                             theta_candidate,
// //                                             phi_e,
// //                                             CHI, 
// //                                             D_candidate);

// //                             if (first_iter){
// //                                 best_y = candidate_y;
// //                                 first_iter = false;
// //                             }
// //                             else if (findmax && best_y < candidate_y){
// //                                 best_y = candidate_y;
// //                             }else if (!findmax && best_y > candidate_y){
// //                                 best_y = candidate_y;
// //                             }
// //                             theta_candidate += theta_stepsize;
// //                         }
// //                     }
// //                 }
// //             }
// //         }
// //     }
// //     else {
// //         // for (int D_idx = 0; D_idx < 2; D_idx ++){
// //             // float D_candidate = D_candidates[D_idx];
// //         float D_candidate = d_upper_bound;

// //         for (int i = 0; i < N; i++){
// //             candidate_y = eqn_54(0.0f, //a_i_candidate,
// //                             f, 
// //                             mu_i, 
// //                             nu_i, 
// //                             X_i_t,
// //                             Y_i_t,
// //                             0.0f, //v_i_candidate, 
// //                             v_e, 
// //                             0.0f, //psi_i_candidate,
// //                             psi_e,
// //                             a_e,
// //                             theta_candidate,
// //                             phi_e,
// //                             CHI, 
// //                             D_candidate);

// //             if (first_iter){
// //                 best_y = candidate_y;
// //                 first_iter = false;
// //             }
// //             else if (findmax && best_y < candidate_y){
// //                 best_y = candidate_y;
// //             }else if (!findmax && best_y > candidate_y){
// //                 best_y = candidate_y;
// //             }
// //             theta_candidate += theta_stepsize;
// //         }
// //         // }
// //     }
// //     return best_y;
// // }


// __global__
// void certify_u_for_nu(
//             float f, 
//             float* X_i_ts, 
//             float* Y_i_ts,
//             float* mu_is,
//             float* nu_is, 
//             float* offsets,
//             float v_e, 
//             float a_e,
//             float omega_e, 
//             float alpha_e, 
//             unsigned int N_points,
//             unsigned int N_pixels,
//             bool* u_certified_for_nu,
//             float* nu_b_out,
//             float* nu_i_out,
//             float* nu_dot_out,
//             int8_t* animateds,
//             int8_t* rois, // area of interest?
//             float* d_upper_bounds,
//             float* d_lower_bounds)
// {
//     int blockId = blockIdx.x + blockIdx.y * gridDim.x;
//     int tid = blockId * (blockDim.x * blockDim.y)
//         + (threadIdx.y * blockDim.x) + threadIdx.x;
    
//     if (tid >= IMG_H * IMG_W){
//         return;
//     } 
    
//     float X_i = X_i_ts[tid];
//     float Y_i = Y_i_ts[tid];
//     float mu_i = mu_is[tid];
//     float nu_i = nu_is[tid]; 
//     float d_upper_bound = d_upper_bounds[tid];
//     float d_lower_bound = d_lower_bounds[tid];
//     float offset = offsets[tid];
//     int8_t animated = animateds[tid];
//     int8_t roi = rois[tid];

//     nu_i_out[tid] = nu_i;
//     bool nu_find_upperbound = Y_i > Y_MIN; // pixel index 360, see whimsicle

//     // if (nu_find_upperbound && nu_i >= nu_b){
//     //     float nu_upper = optimize_nu_dot_i(
//     //         f, 
//     //         mu_i,
//     //         nu_i, 
//     //         X_i_t, 
//     //         Y_i_t, 
//     //         v_e, 
//     //         psi_e, 
//     //         a_e, 
//     //         phi_e, 
//     //         N_points,
//     //         true,
//     //         animated,
//     //         d_upper_bound,
//     //         d_lower_bound);
//     //     u_certified_for_nu[tid] = nu_upper < -offset;
//     //     nu_dot_out[tid] = nu_upper;
//     // } else 
//     if (roi == 0){ // not a target -> skip computation
//         u_certified_for_nu[tid] = true;
//         nu_dot_out[tid] = 0.0f;
//         nu_b_out[tid] = 0.0f;
//     }
//     else if (nu_find_upperbound && nu_i <= NU_B){
//         nu_b_out[tid] = NU_B;
//         float nu_lower = optimize_nu_dot_i2(
//             f, 
//             X_i, 
//             Y_i, 
//             mu_i,
//             nu_i, 
//             v_e, 
//             a_e,
//             omega_e,
//             alpha_e,
//             // unsigned int N,
//             false,
//             animated,
//             d_upper_bound,
//             d_lower_bound);
//         u_certified_for_nu[tid] = nu_lower > 0.0f; //offset;
//         nu_dot_out[tid] = nu_lower;
//     } else {
//         u_certified_for_nu[tid] = true;
//         nu_dot_out[tid] = 0.0f;
//         nu_b_out[tid] = 0.0f;
//     }
// }
